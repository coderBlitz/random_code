#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h" //For the NVIDIA CUDA functionality
// ""


/* File name can be *.cu
   Use developer command prompt, and compile with "nvcc filename.cu"
*/

__global__ void multiplyMtx(int* a,int* b,int* c,int* n)
{
	int id = blockIdx.x;
	if (id < *n){
		c[id] = a[id] * b[id];
	}
}

int main()
{
	const int N = 4;
	int a[N] = {1,4,2,9};
	int b[N] = {8,3,0,6};
	int sum[N];
	
	int* dev_a;
	int* dev_b;
	int* dev_c;
	int* max;

	hipMalloc(&dev_a, N * sizeof(int));
	hipMalloc(&dev_b, N * sizeof(int));
	hipMalloc(&dev_c, N * sizeof(int));
	hipMalloc(&max, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(max, &N, N * sizeof(int), hipMemcpyHostToDevice);

	multiplyMtx <<<N,1>>>(dev_a,dev_b,dev_c,max);

	hipMemcpy(sum, dev_c, N * sizeof(int) ,hipMemcpyDeviceToHost);

	for (int j = 0; j < N; j++){
		printf("Value at product[%d]: %d\n", j, sum[j]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(max);
	hipDeviceReset();
    return 0;
}
