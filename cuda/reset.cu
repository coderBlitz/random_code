
#include "hip/hip_runtime.h"
#include<stdio.h>

int main(){
	hipError_t err;
	err = hipDeviceSynchronize();
	if(err != hipSuccess){
		printf("%s\n",hipGetErrorString(err));
		exit(1);
	}
	err = hipDeviceReset();
	if(err != hipSuccess){
		printf("%s\n",hipGetErrorString(err));
		exit(1);
	}
}
