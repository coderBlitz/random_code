#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdint.h>
#include<sys/time.h>
#include<sys/sysinfo.h>
#include<signal.h>
#include"hip/hip_runtime.h"


/* For starting number N, if N is odd then 3N+1, if N is even N/2
   Program finds chain length for each number, and returns highest
   *Note* The '+2' in multiple places is because I'm skipping 0 and 1 (not anymore)
*/

#define NUM_BLOCKS 5669 // 22725 max using 1 thread. 16069 using 2 (205710954 Bytes of memory to work with)
//#define NUM_BLOCKS 4
#define threads 16 // Try and keep threads powers of 16 (powers of 2 at minimum)

__global__ void chains(int *arr, long TOTAL, uint64_t OFFSET){
	unsigned int UID = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;// 2-D Blocks, 1-D threads
	uint64_t N = UID + OFFSET;// N is the number computed, UIDgives position OFFSET is for multiple runs
	uint64_t count=0; // Running total for compute loop

//	printf("UID: %u\nN: %lu\nOFFSET: %lu\nMAX: %lu\n",UID, N, OFFSET, MAX);
//	if(UID == TOTAL+1) printf("UID: %u\nN: %lu\nOFFSET: %lu\nMAX: %lu\n",UID, N, OFFSET, MAX);

// The only reason I check for this is because the block dimensions dont change with N
// Hence why we only need to check the UID for being to large, because the array size remains the same.
	if (!(UID > TOTAL)){
		while(N > 1){// Calculate
			if (N == 0){
				count = 0;
				break;
			}
			if(N%2 == 0) N /= 2;
			else N = 3*N + 1;
			count++;
		}
		arr[UID] = count;// Store result in array
	}
	__syncthreads();
}

void sig(int a){
	hipDeviceReset();
	printf("Ctrl-c pressed. Exiting..\n");
	exit(1);
}

int main(){
	signal(SIGINT, &sig);
	hipDeviceReset();
	hipError_t err;
//	printf("Ready!\n");

	size_t free_mem,total;
	struct sysinfo s;
	sysinfo(&s);
	uint64_t HOST_MEM = s.freeram;
	hipMemGetInfo(&free_mem,&total);// Get memory stats
//	printf("\nTotal mem: %ld\nFree mem: %ld\n",total,free_mem);// For debug purposes

	struct timeval start, end, diff;

	dim3 blocks(NUM_BLOCKS,NUM_BLOCKS);// Define an NxN 2D grid
	uint64_t TOTAL = NUM_BLOCKS*NUM_BLOCKS*threads;// Simplify total size for each chunk (mostly for GPU)

	// Start loop of sizes
	for(uint64_t N = 1e6;N < 100000000; N += 1000000){
/*		printf("N = ", N);
		fflush(stdout);
		fprintf(stderr, "%lu,", N);
		printf("\b\n");
		fflush(stdout);*/
		printf("%lu,", N);

		// Get start time
		gettimeofday(&start, NULL);

//		uint64_t N = 1800000000;// The number of numbers we compute to (1.84bil uses ~7G of memory)
//		uint64_t N = HOST_MEM/sizeof(int);
		uint64_t count = 0;

		int *len_array = (int *)malloc(N * sizeof(int));// Mem for host array
		if (len_array == NULL){
			printf("N = %lu is too large, not enough memory. Lowering N..\n",N);
			while (len_array == NULL){
				len_array = (int *)malloc(N*sizeof(int));
				N -= N/10;
			}
//			exit(1);
		}

		for(uint64_t i=0;i<N;i++) len_array[i] = 0;
	
		int *dev_array;// Ptr for GPU


		hipMemGetInfo(&free_mem,&total);// Get memory stats
		err = hipMalloc(&dev_array,TOTAL*sizeof(int));// Allocate GPU mem for array
		if(err != hipSuccess){// Error checking
			printf("(GPU) Damn it: '%s'\n",hipGetErrorString(err));
			hipFree(dev_array);

			TOTAL -= (TOTAL*4 - free_mem);// Should give us max memory usage per chunk
			printf("Lowering mems...\nTotal now: %ld\nSize: %ld\n",TOTAL,TOTAL*4);
			err = hipMalloc(&dev_array, TOTAL*sizeof(int));
			if(err != hipSuccess){// Error checking
				printf("(GPU) Damn it again: '%s'\n",hipGetErrorString(err));
				exit(1);
			}
//			cudaFree(dev_array);
		}

//		printf("Range 0-%lu\n", N);
		uint64_t temp = 0, REMAIN = TOTAL;
		for(int i=0;count < N;i++){
//			printf("Run %d\n",i+1);

			temp = count;// Hold previous value
			if ((count+TOTAL) > N){
//				printf("Last run..\n");
				REMAIN = N - count;// Makes sure last copy doesn't try to copy more than host array size
				count = N;// Add last little bit to count
			}

//			printf("Running..\nCOUNT=%lu\n",count);
//			printf("Running..\n");

			err = hipDeviceSynchronize();
			if(err != hipSuccess){
				printf("(GPU) Damn it, kernel broke: '%s'\n",hipGetErrorString(err));
			}

			chains<<<blocks,threads>>>(dev_array, REMAIN, temp);// GPU call (number array, chunk size, offset count)

			err = hipDeviceSynchronize();// Wait for everything to finish
//			cudaGetLastError();
			if(err != hipSuccess){
				printf("(GPU) Damn it, kernel broke: '%s'\n",hipGetErrorString(err));
			}

			count += REMAIN;// We do TOTAL numbers of calculations per run up until N

			err = hipMemcpy(len_array + temp, dev_array, REMAIN*sizeof(int), hipMemcpyDeviceToHost);// Get data back
			if(err != hipSuccess){
				printf("(GPU) Damn it: '%s'\n",hipGetErrorString(err));
				exit(1);
			}

			if (temp > count){ // Should catch overflow
				printf("Overflow, so we're done here\n");
				break;
			}
		
			err = hipDeviceSynchronize();
			if(err != hipSuccess){
				printf("(GPU) Damn it, kernel broke: '%s'\n",hipGetErrorString(err));
			}
		}
//		printf("counting..\n");
		uint64_t max=0;
		for(uint64_t i=0;i<N;i++) if(len_array[i] > len_array[max]) max = i;// Find max

//		printf("Longest chain: %lu with a length of %d\n",max,len_array[max]);// Result

		free(len_array);// Clean up
		hipFree(dev_array);

		gettimeofday(&end, NULL);
		timersub(&end, &start, &diff);

		// This mess is to have a CSV format on stderr but mostly normal format on stdout
/*		printf("\nTotal time: ");
		fflush(stdout);

		fprintf(stderr, "%u.", diff.tv_sec);

		printf("\bs ");
		fflush(stdout);

		fprintf(stderr, "%06u", diff.tv_usec);

		printf("us");
		fflush(stdout);

		fprintf(stderr, "\n");

		printf("\n\n");*/
		// End this nightmare
		printf("%u.%06u\n", diff.tv_sec, diff.tv_usec);
		fflush(stdout);
	} // End N loop


	hipDeviceReset();// Make sure we don't have memory leaks or anything like that
}
