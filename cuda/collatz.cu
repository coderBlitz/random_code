#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<sys/sysinfo.h>
#include<signal.h>
#include"hip/hip_runtime.h"


/* For starting number N, if N is odd then 3N+1, if N is even N/2
   Program finds chain length for each number, and returns highest
   *Note* The '+2' in multiple places is because I'm skipping 0 and 1 (not anymore)
*/

#define NUM_BLOCKS 5669 // 22725 max using 1 thread. 16069 using 2 (205710954 Bytes of memory to work with)
//#define NUM_BLOCKS 4
#define threads 16 // Try and keep threads powers of 16 (powers of 2 at minimum)

__global__ void chains(int *arr, long TOTAL, unsigned long OFFSET){
	unsigned int UID = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;// 2-D Blocks, 1-D threads
	unsigned long N = UID + OFFSET;// N is the number computed, UIDgives position OFFSET is for multiple runs
	unsigned long count=0; // Running total for compute loop

//	printf("UID: %u\nN: %lu\nOFFSET: %lu\nMAX: %lu\n",UID, N, OFFSET, MAX);
//	if(UID == TOTAL+1) printf("UID: %u\nN: %lu\nOFFSET: %lu\nMAX: %lu\n",UID, N, OFFSET, MAX);

// The only reason I check for this is because the block dimensions dont change with N
// Hence why we only need to check the UID for being to large, because the array size remains the same.
	if (!(UID > TOTAL)){
		while(N > 1){// Calculate
			if (N == 0){
				count = 0;
				break;
			}
			if(N%2 == 0) N /= 2;
			else N = 3*N + 1;
			count++;
		}
		arr[UID] = count;// Store result in array
	}
	__syncthreads();
}

void sig(int a){
	hipDeviceReset();
	printf("Ctrl-c pressed. Exiting..\n");
	exit(1);
}

int main(){
	signal(SIGINT, &sig);
	hipDeviceReset();
	hipError_t err;
	printf("Ready!\n");

	size_t free_mem,total;
	struct sysinfo s;
	sysinfo(&s);
	unsigned long HOST_MEM = s.freeram;
	hipMemGetInfo(&free_mem,&total);// Get memory stats
//	printf("\nTotal mem: %ld\nFree mem: %ld\n",total,free_mem);// For debug purposes

	dim3 blocks(NUM_BLOCKS,NUM_BLOCKS);// Define an NxN 2D grid
	long TOTAL = NUM_BLOCKS*NUM_BLOCKS*threads;// Simplify total size for each chunk (mostly for GPU)
//	unsigned long N = 1800000000;// The number of numbers we compute to (1.84bil uses ~7G of memory)
	unsigned long N = HOST_MEM/sizeof(int);
	unsigned long count = 0;

	int *len_array = (int *)malloc(N * sizeof(int));// Mem for host array
	if (len_array == NULL){
		printf("N = %lu is too large, not enough memory. Lowering N..\n",N);
		while (len_array == NULL){
			len_array = (int *)malloc(N*sizeof(int));
			N -= N/10;
		}
//		exit(1);
	}

	for(unsigned long i=0;i<N;i++) len_array[i] = 0;
	
	int *dev_array;// Ptr for GPU


	hipMemGetInfo(&free_mem,&total);// Get memory stats
	err = hipMalloc(&dev_array,TOTAL*sizeof(int));// Allocate GPU mem for array
	if(err != hipSuccess){// Error checking
		printf("(GPU) Damn it: '%s'\n",hipGetErrorString(err));
		hipFree(dev_array);

		TOTAL -= (TOTAL*4 - free_mem);// Should give us max memory usage per chunk
		printf("Lowering mems...\nTotal now: %ld\nSize: %ld\n",TOTAL,TOTAL*4);
		err = hipMalloc(&dev_array, TOTAL*sizeof(int));
		if(err != hipSuccess){// Error checking
			printf("(GPU) Damn it again: '%s'\n",hipGetErrorString(err));
			exit(1);
		}
//		cudaFree(dev_array);
	}

	printf("Range 0-%lu\n", N);
	unsigned long temp = 0, REMAIN = TOTAL;
	for(int i=0;count < N;i++){
		printf("Run %d\n",i+1);

		temp = count;// Hold previous value
		if ((count+TOTAL) > N){
			printf("Finishing last run..\n");
			REMAIN = N - count;// Makes sure last copy doesn't try to copy more than host array size
			count = N;// Add last little bit to count
		}

//	printf("Running..\nCOUNT=%lu\n",count);
	printf("Running..\n");

		err = hipDeviceSynchronize();
		if(err != hipSuccess){
			printf("(GPU) Damn it, kernel broke: '%s'\n",hipGetErrorString(err));
		}

		chains<<<blocks,threads>>>(dev_array, REMAIN, temp);// GPU call (number array, chunk size, offset count)

		err = hipDeviceSynchronize();// Wait for everything to finish
//		cudaGetLastError();
		if(err != hipSuccess){
			printf("(GPU) Damn it, kernel broke: '%s'\n",hipGetErrorString(err));
		}

		count += REMAIN;// We do TOTAL numbers of calculations per run up until N

		err = hipMemcpy(len_array + temp, dev_array, REMAIN*sizeof(int), hipMemcpyDeviceToHost);// Get data back
		if(err != hipSuccess){
			printf("(GPU) Damn it: '%s'\n",hipGetErrorString(err));
			exit(1);
		}

		if (temp > count){ // Should catch overflow
			printf("Overflow, so we're done here\n");
			break;
		}
		
		err = hipDeviceSynchronize();
		if(err != hipSuccess){
			printf("(GPU) Damn it, kernel broke: '%s'\n",hipGetErrorString(err));
		}
	}
	printf("counting..\n");
	unsigned long max=0;
	for(unsigned long i=0;i<N;i++) if(len_array[i] > len_array[max]) max = i;// Find max

//for(int i=268549000;i<268550000;i++) printf("%d- %lu\n",i,len_array[i]);// Debug print
	printf("Longest chain: %lu with a length of %d\n",max,len_array[max]);// Result
	
//	for(int i=0;i<N;i++) printf("len_arr[%ld]: %ld\n",i,len_array[i]);// Alternate debug print

	free(len_array);// Clean up
	hipFree(dev_array);
	hipDeviceReset();// Make sure we don't have memory leaks or anything like that
}
